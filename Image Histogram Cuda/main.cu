#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "scrImagePgmPpmPackage.cuh"

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32

using namespace std;

// calculates the resized image
__global__ void calculateHistogram(unsigned int* imageHistogram, unsigned int width, unsigned int height, hipTextureObject_t texObj)
{
	const unsigned int tidX = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int tidY = blockIdx.y * blockDim.y + threadIdx.y;

	const unsigned int localId = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned int histStartIndex = (blockIdx.y * gridDim.x + blockIdx.x) * 256;

	__shared__ unsigned int histo_private[256];

	if (localId < 256)
	{
		histo_private[localId] = 0;
	}
	__syncthreads();

	// step 4: read the texture memory from the texture object
	unsigned char imageData = tex2D<unsigned char>(texObj, (float)(tidX), (float)(tidY));
	atomicAdd(&(histo_private[imageData]), 1);

	__syncthreads();

	if (localId < 256)
	{
		imageHistogram[histStartIndex + localId] = histo_private[localId];
	}
}

int main()
{
	int height = 0;
	int width = 0;
	int nHistogram = 0;

	// define the scaling ratio
	unsigned char* data;
	unsigned int* imageHistogram, * d_imageHistogram;

	char inputStr[1024] = { "" };
	hipError_t returnValue;

	// create a channel description to be used while linking the texture object
	hipArray* cu_array;
	hipChannelFormatKind kind = hipChannelFormatKindUnsigned;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, kind);

	get_PgmPpmParams(inputStr, &height, &width); // get height and width of image
	data = (unsigned char*)malloc(height * width * sizeof(unsigned char));
	printf("Reading image of width (%d) and height (%d)\n", width, height);
	scr_read_pgm(inputStr, data, height, width); // load image data to "data"

	// one histogram per image block. Size range of 0-255 since char image.
	nHistogram = (height / BLOCK_SIZE_Y) * (width / BLOCK_SIZE_X) * 256;
	imageHistogram = (unsigned int*)malloc(nHistogram * sizeof(unsigned int));

	// allocate CUDA array
	returnValue = hipMallocArray(&cu_array, &channelDesc, width, height);
	if (returnValue != hipSuccess)
	{
		printf("Error while running CUDA API Malloc Array\n");
	}

	returnValue = hipMemcpy(cu_array, data, height * width * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (returnValue != hipSuccess)
	{
		printf("Error while running CUDA API Array Copy Host to Device\n");
	}

	// step 1: specify textures
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cu_array;

	// step 2: specify texture object parameters
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	// step 3: create texture object
	hipTextureObject_t texObj;
	returnValue = hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
	
	if (returnValue != hipSuccess)
	{
		printf("Error while running CUDA API Bind Texture\n");
	}

	returnValue = hipMalloc(&d_imageHistogram, nHistogram * sizeof(unsigned int));

	if (returnValue != hipSuccess)
	{
		printf("Error while running CUDA API Malloc\n");
	}

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
	dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);
	printf("Launching grid with blocks (%d)(%d)", dimGrid.x, dimGrid.y);

	calculateHistogram << <dimGrid, dimBlock >> > (d_imageHistogram, width, height, texObj);

	returnValue = hipDeviceSynchronize();
	if (returnValue != hipSuccess)
	{
		printf("Error while running CUDA API kernel\n");
	}

	returnValue = hipMemcpy(imageHistogram, d_imageHistogram, nHistogram * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (returnValue != hipSuccess)
	{
		printf("Error while running CUDA API Memcpy Device to Host\n");
	}

	// step 5: destroy texture object
	hipDestroyTextureObject(texObj);

	printf("Histogram per section is as follows: ");
	for (int i = 0; i < nHistogram / 256; i++)
	{
		printf("---------------------- Histogram for block %d ----------------------\n", i);
		for (int j = 0; j < 256; j++)
		{
			int index = i * 256 + j;
			printf("[%d=[%d]] ", j, imageHistogram[index]);
		}
		printf("\n");
	}

	if (data != NULL)
	{
		free(data);
	}

	if (cu_array != NULL)
	{
		hipFreeArray(cu_array);
	}

	if (imageHistogram != NULL)
	{
		free(imageHistogram);
	}

	if (d_imageHistogram != NULL)
	{
		hipFree(d_imageHistogram);
	}

	return 0;
}
